#include "hip/hip_runtime.h"
#include <vector>
#include <stdlib.h> /* srand, rand */
#include <boost/filesystem.hpp>

#include "caffe/filler.hpp"
#include "caffe/layers/repartition_layer.hpp"
#include "caffe/util/interp.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/more_math_functions.hpp"

namespace caffe {

template <typename Dtype>
cv::Scalar gray2jet(Dtype f) {

  /*plot short rainbow RGB*/
  Dtype a = f / 0.25;            // invert and group
  int X = floor(a);              // this is the integer part
  int Y = floor(255 * (a - X));  // fractional part from 0 to 255
  int Z = floor(128 * (a - X));  // fractional part from 0 to 128
  int r, g, b;
  switch (X) {
    case 0:
      r = 0;
      g = Y;
      b = 128 - Z;
      break;
    case 1:
      r = Y;
      g = 255;
      b = 0;
      break;
    case 2:
      r = 255;
      g = 255 - Z;
      b = 0;
      break;
    case 3:
      r = 255;
      g = 128 - Z;
      b = 0;
      break;
    case 4:
      r = 255;
      g = 0;
      b = 0;
      break;
  }
  // opencv is bgr, not rgb
  return cv::Scalar(b, g, r);
}

template <typename Dtype>
Dtype GetDtypePrecision(Dtype value, Dtype precision) {
  return (floor((value * pow(10, precision) + 0.5)) / pow(10, precision));
}

template <typename Dtype>
void Show_rois(Blob<Dtype> *rois_blob, Blob<Dtype> *filter_blob,
               Blob<Dtype> *label_blob, const int save_id, const int num_im,
               const vector<string> voc_label, const int ignore_label,
               const float predict_threshold, const bool jet = false) {
  const int num_roi = rois_blob->num();
  const int num_class = filter_blob->channels();
  const Dtype *rois = rois_blob->cpu_data();
  const Dtype *filter = filter_blob->cpu_data();
  const Dtype *label = label_blob->cpu_data();
  const int each_page_num = 1;
  const int line_width = 6;

  cv::RNG rng(12345);
  stringstream save_path;
  stringstream save_dir;

  // load new image
  stringstream load_path;
  load_path << "tmp/" << save_id << "_.png";
  LOG(INFO) << "load png path: " << load_path.str();
  cv::Mat im_mat = cv::imread(load_path.str());
  cv::Mat im_mat_o = im_mat.clone();
  const int width_im = im_mat.cols;
  const int height_im = im_mat.rows;

  for (int c = 0; c < num_class; ++c) {
    if (label[c] <= predict_threshold) {
      continue;
    }
    if (c == ignore_label) {
      continue;
    }

    save_dir.str(std::string());
    save_dir << "tmp/" << voc_label[c] << "/" << save_id << "/";
    boost::filesystem::create_directories(save_dir.str());

    cv::Mat add_mat(im_mat.rows, im_mat.cols, CV_32FC1);
    cv::Mat count_mat(im_mat.rows, im_mat.cols, CV_32FC1, cv::Scalar(0));
    cv::Mat score_mat(im_mat.rows, im_mat.cols, CV_32FC1, cv::Scalar(0));
    cv::Mat norma_mat(im_mat.rows, im_mat.cols, CV_32FC1, cv::Scalar(0));
    for (int r = 0, page = 0; r < num_roi; ++r) {
      // rois: n x1 y1 x2 y2
      // rec: x y w h
      //

      Dtype wstart = round(rois[5 * r + 1]);
      Dtype hstart = round(rois[5 * r + 2]);
      Dtype wend = round(rois[5 * r + 3]);
      Dtype hend = round(rois[5 * r + 4]);

      // caculate the inner and outer RoI coordinate
      Dtype width_roi = wend - wstart;
      Dtype height_roi = hend - hstart;
      Dtype context_scale = 1.8;
      // Dtype context_scale = sqrtf(2.0);
      Dtype width_roi_inner = width_roi / context_scale;
      Dtype height_roi_inner = height_roi / context_scale;
      Dtype width_roi_outer = width_roi * context_scale;
      Dtype height_roi_outer = height_roi * context_scale;
      Dtype wcenter = (wend + wstart) / 2.0;
      Dtype hcenter = (hend + hstart) / 2.0;

      Dtype wstart_inner = wcenter - width_roi_inner / 2.0;
      Dtype hstart_inner = hcenter - height_roi_inner / 2.0;
      Dtype wend_inner = wcenter + width_roi_inner / 2.0;
      Dtype hend_inner = hcenter + height_roi_inner / 2.0;

      Dtype wstart_outer = max(wcenter - width_roi_outer / 2.0, 0.0);
      Dtype hstart_outer = max(hcenter - height_roi_outer / 2.0, 0.0);
      Dtype wend_outer = min(wcenter + width_roi_outer / 2.0, width_im * 1.0);
      Dtype hend_outer = min(hcenter + height_roi_outer / 2.0, height_im * 1.0);

      cv::Rect rec = cv::Rect(wstart, hstart, wend - wstart, hend - hstart);
      cv::Rect rec_inner =
          cv::Rect(wstart_inner, hstart_inner, wend_inner - wstart_inner,
                   hend_inner - hstart_inner);
      cv::Rect rec_outer =
          cv::Rect(wstart_outer, hstart_outer, wend_outer - wstart_outer,
                   hend_outer - hstart_outer);

      Dtype rois_score = filter[r * num_class + c];
      //--------------------------------------------------------------------------
      // draw rectangle
      cv::rectangle(im_mat_o, rec, gray2jet(abs(rois_score)), line_width);
      cv::rectangle(im_mat_o, rec_inner, gray2jet(abs(rois_score)), line_width);
      cv::rectangle(im_mat_o, rec_outer, gray2jet(abs(rois_score)), line_width);

      // 如果r+1整除each_page_num或者r是最后一个
      if ((r + 1) % each_page_num == 0 || r == num_roi - 1) {
        save_path.str(std::string());
        save_path.precision(4);
        save_path << save_dir.str() << (rois_score > 0 ? "+" : "-")
                  << std::fixed << abs(rois_score) << "_" << page << ".png";
        cv::imwrite(save_path.str(), im_mat_o);
        LOG(INFO) << "save_path: " << save_path.str();

        page++;
        im_mat.copyTo(im_mat_o);
      }

      //--------------------------------------------------------------------------
      // 计算 heat map
      cv::Mat mask_mat(im_mat.rows, im_mat.cols, CV_8UC1, cv::Scalar(0));
      cv::Mat roi_mat = mask_mat(rec);
      roi_mat = 1;

      add_mat = 1;
      // CV_32FC1 CV_32FC1 CV_32FC1 CV_8UC1
      cv::add(count_mat, add_mat, count_mat, mask_mat);

      add_mat = rois_score > 0 ? rois_score : 0;
      // CV_32FC1 CV_32FC1 CV_32FC1 CV_8UC1
      cv::add(score_mat, add_mat, score_mat, mask_mat);
    }

    //----------------------------------------------------------------------------
    if (!jet) continue;
    double maxVal, minVal, alpha, beta;
    cv::Mat u8_mat;
    cv::Mat cm_mat;

    // 保存 count map
    cv::minMaxLoc(count_mat, &minVal, &maxVal);
    alpha = 255.0 / (maxVal - minVal);
    beta = alpha * minVal;
    LOG(INFO) << "maxVal: " << maxVal << " minVal: " << minVal;

    count_mat.convertTo(u8_mat, CV_8UC1, alpha, beta);
    save_path.str(std::string());
    save_path << save_dir.str() << "rois_c.png";
    cv::imwrite(save_path.str(), u8_mat);

    /*cv::applyColorMap(u8_mat, cm_mat, cv::COLORMAP_JET);*/
    /*save_path.str(std::string());*/
    /*save_path << save_dir.str() << "rois_cj.png";*/
    /*cv::imwrite(save_path.str(), cm_mat);*/

    // 保存 score map
    cv::minMaxLoc(score_mat, &minVal, &maxVal);
    alpha = 255.0 / (maxVal - minVal);
    beta = alpha * minVal;
    LOG(INFO) << "maxVal: " << maxVal << " minVal: " << minVal;

    score_mat.convertTo(u8_mat, CV_8UC1, alpha, beta);
    save_path.str(std::string());
    save_path << save_dir.str() << "rois_s.png";
    cv::imwrite(save_path.str(), u8_mat);

    // cv::applyColorMap(u8_mat, cm_mat, cv::COLORMAP_JET);
    // save_path.str(std::string());
    // save_path << save_dir.str() << "rois_sj.png";
    // cv::imwrite(save_path.str(), cm_mat);

    // 保存 norm map
    cv::divide(score_mat, count_mat, norma_mat);
    cv::minMaxLoc(norma_mat, &minVal, &maxVal);
    alpha = 255.0 / (maxVal - minVal);
    beta = alpha * minVal;
    LOG(INFO) << "maxVal: " << maxVal << " minVal: " << minVal;

    norma_mat.convertTo(u8_mat, CV_8UC1, alpha, beta);
    save_path.str(std::string());
    save_path << save_dir.str() << "rois_n.png";
    cv::imwrite(save_path.str(), u8_mat);

    // cv::applyColorMap(u8_mat, cm_mat, cv::COLORMAP_JET);
    // save_path.str(std::string());
    // save_path << save_dir.str()<< "rois_nj.png";
    // cv::imwrite(save_path.str(), cm_mat);
  }
}

template <typename Dtype>
void Show_blob(const Dtype *data, const int channels, const int height,
               const int width, const string save_opg_path,
               const float threshold_, const int fill = 0) {
  Dtype maxval = -FLT_MAX;
  Dtype sum = 0;
  for (int c = 0; c < channels; c++) {
    for (int h = 0; h < height; h++) {
      for (int w = 0; w < width; w++) {
        int index = (c * height + h) * width + w;
        /*Dtype value = abs(data[index]);*/
        Dtype value = data[index] > 0 ? data[index] : 0;
        if (value > maxval) {
          maxval = value;
        }
        sum += value;
      }
    }
  }
  LOG(INFO) << "raw mean: " << sum / channels / height / width;

  if (threshold_ > 0) {
    maxval = maxval * threshold_;
  } else {
    maxval = sum / channels / height / width;
  }
  Dtype scale_factor = 255.0 / maxval;

  //-----------------------------------------------------------------------
  cv::Mat opg_mat;
  if (channels == 3) {
    opg_mat = cv::Mat(height, width, CV_8UC3);
  } else if (channels == 1) {
    opg_mat = cv::Mat(height, width, CV_8UC1);
  } else {
    LOG(FATAL) << "channels should 1 or 3";
  }

  sum = 0;
  uchar *opg_mat_data = opg_mat.data;
  for (int c = 0; c < channels; c++) {
    for (int h = 0; h < height; h++) {
      for (int w = 0; w < width; w++) {
        int index = (c * height + h) * width + w;
        int index_mat = (h * width + w) * channels + c;
        /*Dtype value = abs(data[index]);*/
        Dtype value = data[index] > 0 ? data[index] : 0;
        if (value > maxval) {
          opg_mat_data[index_mat] = 255;
          sum += maxval;
        } else {
          if (fill >= 0) {
            opg_mat_data[index_mat] = fill;
          } else {
            opg_mat_data[index_mat] = scale_factor * value;
          }
          sum += value;
        }
      }
    }
  }

  LOG(INFO) << "max_value: " << maxval
            << " mean: " << sum / channels / height / width;

  cv::imwrite(save_opg_path, opg_mat);

  //-----------------------------------------------------------------------
  /*const Dtype* opg_cpu=opg_blob->cpu_data();*/
  /*int total[26];*/
  /*for(int i=0;i<26;++i){*/
  /*total[i]=0;*/
  /*}*/
  /*for(int e=0;e<opg_blob->count();e++){*/
  /*int level=int(opg_cpu[e]/10);*/
  /*total[level]++;*/
  /*}*/
  /*for(int i=0;i<26;++i){*/
  /*std::cout << i<<":"<<total[i]<<" ";*/
  /*}*/
  /*std::cout<<std::endl;*/
  //-----------------------------------------------------------------------
}

template <typename Dtype>
bool RepartitionLayer<Dtype>::aou_small(const Dtype *roi,
                                        const Dtype bb_offset) {
  // id x1 y1 x2 y2
  const int bbox_num = bboxes_->shape(0);
  for (int b = 0; b < bbox_num; ++b) {
    const Dtype *bbox = bboxes_->cpu_data() + bboxes_->offset(b);
    if (bbox[0] != roi[0]) continue;

    // contain
    if (bbox[1] >= roi[1] - bb_offset && bbox[2] >= roi[2] - bb_offset &&
        bbox[3] <= roi[3] + bb_offset && bbox[4] <= roi[4] + bb_offset)
      return true;

    Dtype ow = std::min(bbox[3], roi[3]) - std::max(bbox[1], roi[1]) + 1;
    Dtype oh = std::min(bbox[4], roi[4]) - std::max(bbox[2], roi[2]) + 1;

    if (ow <= 0 || oh <= 0) continue;

    Dtype ov = ow * oh;
    Dtype s_b = (bbox[3] - bbox[1] + 1) * (bbox[4] - bbox[2] + 1);

    /*float
     * aou=ov/((bbox[2]-bbox[0])*(bbox[3]-bbox[1])+(roi[2]-roi[0])*(roi[3]-roi[1])-ov);*/
    Dtype aou = ov / s_b;
    if (aou > 0.7) return true;
  }

  return false;
}

template <typename Dtype>
__global__ void get_above_mask(const int N, const Dtype *const data,
                               Dtype *const mask, const Dtype threshold) {
  CUDA_KERNEL_LOOP(index, N) {
    if (data[index] >= threshold)
      mask[index] = 1;
    else
      mask[index] = 0;
  }
}

template <typename Dtype>
void RepartitionLayer<Dtype>::Score_map_crf() {
  crf_data_->ReshapeLike(*raw_data_);
  caffe_copy(crf_data_->count(), raw_data_->cpu_data(),
             crf_data_->mutable_cpu_data());

  //-----------------------------------------------------------------------
  const vector<string> layer_names;
  //= net_->layer_names();
  // int conv4_index = -1;
  // int conv5_index = -1;
  for (size_t i = 0; i < layer_names.size(); i++) {
    if (layer_names[i].compare("conv4_3") == 0) {
      // conv4_index = i;
    }
    if (layer_names[i].compare("conv5_3") == 0) {
      // conv5_index = i;
    }
  }

  const vector<int> conv4_bottom_ids;
  //= net_->bottom_ids(conv4_index);
  shared_ptr<Blob<Dtype> > conv4_blob;
  //= net_->blobs()[conv4_bottom_ids[0]];
  const vector<int> conv5_bottom_ids;
  //= net_->bottom_ids(conv5_index);
  shared_ptr<Blob<Dtype> > conv5_blob;
  //= net_->blobs()[conv5_bottom_ids[0]];

  shared_ptr<Blob<Dtype> > feature_blob = conv5_blob;

  //-----------------------------------------------------------------------
  vector<int> crf_opg_shape = raw_data_->shape();
  crf_opg_shape[1] = 2;
  crf_opg_->Reshape(crf_opg_shape);
  caffe_set(crf_opg_->count(), Dtype(0), crf_opg_->mutable_cpu_data());

  Blob<Dtype> fusion_blob;
  vector<int> fusion_shape = feature_blob->shape();
  fusion_shape[1] = 1;
  fusion_blob.Reshape(fusion_shape);
  caffe_set(fusion_blob.count(), Dtype(0), fusion_blob.mutable_cpu_data());

  const int a_offset = fusion_blob.offset(0, 1, 0, 0);
  for (int c = 0; c < feature_blob->channels(); ++c) {
    /*caffe_abs(a_offset, feature_blob->cpu_data() + c * a_offset,
     * crf_opg_->mutable_cpu_diff());*/
    /*caffe_add(a_offset, crf_opg_->cpu_diff(), crf_opg_->cpu_data(),
     * crf_opg_->mutable_cpu_data());*/
    caffe_add(a_offset, feature_blob->cpu_data() + c * a_offset,
              fusion_blob.cpu_data(), fusion_blob.mutable_cpu_data());
  }

  caffe_cpu_interp2<Dtype, false>(
      1, fusion_blob.cpu_data(), 0, 0, fusion_shape[2], fusion_shape[3],
      fusion_shape[2], fusion_shape[3], crf_opg_->mutable_cpu_data(), 0, 0,
      crf_opg_shape[2], crf_opg_shape[3], crf_opg_shape[2], crf_opg_shape[3]);

  const Dtype max_value = max_element_(crf_opg_->cpu_data(), crf_opg_->count());
  const Dtype scale_factor = 1 / (max_value);
  crf_opg_->scale_data(scale_factor);
  Dtype *crf_opg = crf_opg_->mutable_cpu_data();
  for (int i = 0; i < crf_opg_->count(); ++i) {
    if (crf_opg[i] < 0.0) {
      crf_opg[i] = 0;
    }
  }

  if (debug_info_) {
    LOG(INFO) << "max_value: " << (max_value);
  }

  //-----------------------------------------------------------------------
  crf_data_dim_->Reshape(1, 2, 1, 1);
  crf_data_dim_->mutable_cpu_data()[0] = raw_data_->shape(2);
  crf_data_dim_->mutable_cpu_data()[1] = raw_data_->shape(3);

  //-----------------------------------------------------------------------
  crf_layer_->Forward(crf_bottom_vec_, crf_top_vec_);

  stringstream save_path;
  save_path << "tmp/" << pass_im_ << "_feat.png";
  /*Show_blob(crf_output, false, n, rows, cols, channels,
   * save_crf_opg_path.str());*/
  Show_blob(crf_output_->cpu_data(), 1, crf_output_->height(),
            crf_output_->width(), save_path.str(), 1);

  stringstream save_fusion_path;
  save_fusion_path << "tmp/" << pass_im_ << "_fusion.png";
  Show_blob(crf_opg_->cpu_data(), 1, crf_opg_->height(), crf_opg_->width(),
            save_fusion_path.str(), 1);
}

template <typename Dtype>
void RepartitionLayer<Dtype>::Repartition_crf(const int label) {
  const vector<int> start_bottom_ids;
  //= net_->bottom_ids(start_index_);
  shared_ptr<Blob<Dtype> > im_blob;
  //= net_->blobs()[start_bottom_ids[0]];

  crf_data_->ReshapeLike(*im_blob);
  caffe_copy(crf_data_->count(), im_blob->cpu_data(),
             crf_data_->mutable_cpu_data());

  vector<int> opg_shape = im_blob->shape();
  opg_shape[1] = 2;
  crf_opg_->Reshape(opg_shape);

  /*caffe_copy(crf_opg_->count(), im_blob->cpu_diff(),*/
  /*crf_opg_->mutable_cpu_data());*/

  const int a_offset = crf_opg_->offset(0, 1, 0, 0);
  caffe_abs(a_offset, im_blob->cpu_diff(), crf_opg_->mutable_cpu_data());
  caffe_abs(a_offset, im_blob->cpu_diff() + 1 * a_offset,
            crf_opg_->mutable_cpu_diff());
  caffe_add(a_offset, crf_opg_->cpu_diff(), crf_opg_->cpu_data(),
            crf_opg_->mutable_cpu_data());
  caffe_abs(a_offset, im_blob->cpu_diff() + 2 * a_offset,
            crf_opg_->mutable_cpu_diff());
  caffe_add(a_offset, crf_opg_->cpu_diff(), crf_opg_->cpu_data(),
            crf_opg_->mutable_cpu_data());

  const Dtype *max_value = std::max_element(
      crf_opg_->cpu_data(), crf_opg_->cpu_data() + crf_opg_->count());

  if (debug_info_) {
    LOG(INFO) << "Repartition_crf max_value: " << *max_value;
  }
  const Dtype scale_factor = 1 / (*max_value);
  crf_opg_->scale_data(scale_factor);
  Dtype *crf_opg = crf_opg_->mutable_cpu_data();
  for (int i = 0; i < crf_opg_->count(); ++i) {
    if (crf_opg[i] < crf_threshold_) {
      crf_opg[i] = 0;
    }
  }

  /*caffe_cpu_axpby(a_offset, Dtype(-1), crf_opg_->cpu_data(), Dtype(0),
   * crf_opg_->mutable_cpu_data() + a_offset);*/
  /*caffe_add_scalar(a_offset, Dtype(1), crf_opg_->mutable_cpu_data() +
   * a_offset);*/

  crf_data_dim_->Reshape(1, 2, 1, 1);
  crf_data_dim_->mutable_cpu_data()[0] = im_blob->shape(2);
  crf_data_dim_->mutable_cpu_data()[1] = im_blob->shape(3);

  crf_layer_->Forward(crf_bottom_vec_, crf_top_vec_);

  stringstream save_crf_opg_path;
  save_crf_opg_path << "tmp/" << pass_im_ << "_" << voc_label_[label]
                    << "_crf.png";
  /*Show_blob(crf_output, false, n, rows, cols, channels,
   * save_crf_opg_path.str());*/
  Show_blob(crf_output_->cpu_data(), 1, crf_output_->height(),
            crf_output_->width(), save_crf_opg_path.str(), 1);
}

template <typename Dtype>
__global__ void InitFilter_Test(const int count, const Dtype *const label_data,
                                const int num_class, Dtype *const top_data) {
  CUDA_KERNEL_LOOP(index, count) {
    const int c = index % num_class;
    // TODO(YH): What is the correct threshold in test
    if (label_data[c] > 0.00001) {
      top_data[index] = 1;
    } else {
      top_data[index] = 0;
    }
  }
}

template <typename Dtype>
void RepartitionLayer<Dtype>::InitFilter(const Dtype *const label_gpu_data,
                                         Dtype *const filter_gpu_data) {

  switch (this->layer_param_.cpg_param().mode()) {
    case CPGParameter_Mode_DEFAULT:
    case CPGParameter_Mode_PRED:
    case CPGParameter_Mode_CPG_POOLING:
      if (this->phase_ == TRAIN) {
        caffe_gpu_set(num_roi_ * num_class_, Dtype(1), filter_gpu_data);
      } else {
        // NOLINT_NEXT_LINE(whitespace/operators)
        InitFilter_Test<Dtype> << <CAFFE_GET_BLOCKS(num_roi_ * num_class_),
                                   CAFFE_CUDA_NUM_THREADS>>>
            (num_roi_ * num_class_, label_gpu_data, num_class_,
             filter_gpu_data);
      }
      break;
    case CPGParameter_Mode_CRF:
      break;
    default:
      LOG(FATAL) << "Unknown mode.";
  }
}

template <typename Dtype>
__global__ void ScoreBBoxes(const int num_roi, const Dtype *const opg_data,
                            const int num, const int channels, const int height,
                            const int width, const Dtype *const rois_data,
                            const int num_class, const int label,
                            const Dtype threshold, const Dtype min_density,
                            const Dtype all_mass, Dtype *const top_data,
                            const int r = 5) {
  CUDA_KERNEL_LOOP(index, num_roi) {
    const int rois_index = index;

    const Dtype *const roi = rois_data + 5 * rois_index;
    const int wstart = max(int(roi[1]), 0);
    const int hstart = max(int(roi[2]), 0);
    const int wend = min(int(roi[3]), width);
    const int hend = min(int(roi[4]), height);

    /*Dtype sum = 0;*/
    /*Dtype maxval = -FLT_MAX;*/
    Dtype mass = 0;
    for (int c = 0; c < channels; ++c) {
      const Dtype *gradient = opg_data + c * height * width;
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          /*sum += gradient[h * width + w];*/
          /*sum += exp(max(r * gradient[h * width + w], kLOG_THRESHOLD));*/
          /*if (maxval < gradient[h * width + w]) {*/
          /*maxval = gradient[h * width + w];*/
          /*}*/
          if (threshold < gradient[h * width + w]) {
            mass++;
          }
        }
      }
    }
    Dtype s = (hend - hstart) * (wend - wstart);
    Dtype density = 1.0 * mass / s / channels;
    top_data[rois_index * num_class + label] = density + 1.0 * mass / all_mass;
  }
}

template <typename Dtype>
__global__ void WeightBBoxes(const int num_roi, const Dtype *const opg_data,
                             const int num, const int channels,
                             const int height, const int width,
                             const Dtype *const rois_data, const int num_class,
                             const int cls_id, const Dtype threshold,
                             const Dtype min_density, const Dtype min_mass,
                             Dtype *const top_data, const int r = 5) {
  CUDA_KERNEL_LOOP(index, num_roi) {
    const int rois_index = index;

    const Dtype *const roi = rois_data + 5 * rois_index;
    const int wstart = max(int(roi[1]), 0);
    const int hstart = max(int(roi[2]), 0);
    const int wend = min(int(roi[3]), width);
    const int hend = min(int(roi[4]), height);

    Dtype mass = 0;
    for (int c = 0; c < channels; ++c) {
      const Dtype *gradient = opg_data + c * height * width;
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          if (threshold < gradient[h * width + w]) {
            mass++;
          }
        }
      }
    }
    Dtype s = (hend - hstart) * (wend - wstart);
    Dtype density = 1.0 * mass / s / channels;
    if (density > min_density && mass > min_mass) {
      top_data[rois_index * num_class + cls_id] = Dtype(1);
    } else {
      top_data[rois_index * num_class + cls_id] = Dtype(0);
    }
  }
}

template <typename Dtype>
__global__ void LabelBBoxes_softmax(
    const int num_roi, const Dtype *const opg_data, const int num,
    const int channels, const int height, const int width,
    const Dtype *const rois_data, const int num_class, const int cls_id,
    const Dtype threshold, const Dtype min_density, const Dtype min_mass,
    Dtype *const top_data, const int r = 5) {
  CUDA_KERNEL_LOOP(index, num_roi) {
    const int rois_index = index;

    const Dtype *const roi = rois_data + 5 * rois_index;
    const int wstart = max(int(roi[1]), 0);
    const int hstart = max(int(roi[2]), 0);
    const int wend = min(int(roi[3]), width);
    const int hend = min(int(roi[4]), height);

    Dtype mass = 0;
    for (int c = 0; c < channels; ++c) {
      const Dtype *gradient = opg_data + c * height * width;
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          if (threshold < gradient[h * width + w]) {
            mass++;
          }
        }
      }
    }
    Dtype s = (hend - hstart) * (wend - wstart);
    Dtype density = 1.0 * mass / s / channels;
    if (density > min_density && mass > min_mass) {
      top_data[rois_index * num_class + cls_id] = Dtype(cls_id);
    } else {
      top_data[rois_index * num_class + cls_id] = Dtype(num_class - 1);
    }
  }
}

template <typename Dtype>
__global__ void LabelBBoxes(const int num_roi, const Dtype *const opg_data,
                            const int num, const int channels, const int height,
                            const int width, const Dtype *const rois_data,
                            const int num_class, const int cls_id,
                            const Dtype threshold, const Dtype min_density,
                            const Dtype min_mass, Dtype *const top_data,
                            const int r = 5) {
  CUDA_KERNEL_LOOP(index, num_roi) {
    const int rois_index = index;

    const Dtype *const roi = rois_data + 5 * rois_index;
    const int wstart = max(int(roi[1]), 0);
    const int hstart = max(int(roi[2]), 0);
    const int wend = min(int(roi[3]), width);
    const int hend = min(int(roi[4]), height);

    Dtype mass = 0;
    for (int c = 0; c < channels; ++c) {
      const Dtype *gradient = opg_data + c * height * width;
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          if (threshold < gradient[h * width + w]) {
            mass++;
          }
        }
      }
    }
    Dtype s = (hend - hstart) * (wend - wstart);
    Dtype density = 1.0 * mass / s / channels;
    if (density > min_density && mass > min_mass) {
      top_data[rois_index * num_class + cls_id] = Dtype(1);
    } else {
      top_data[rois_index * num_class + cls_id] = Dtype(0);
    }
  }
}

template <typename Dtype>
__global__ void CPGPooling(const int num_roi, const Dtype *opg_data,
                           const int num, const int channels,
                           const int height_im, const int width_im,
                           const Dtype *rois_data, const int num_class,
                           const int cls_id, const Dtype threshold,
                           const Dtype min_density, const Dtype min_mass,
                           Dtype *const top_data) {
  CUDA_KERNEL_LOOP(index, num_roi) {
    int rois_index = index;

    rois_data += 5 * rois_index;
    Dtype wstart = round(rois_data[1]);
    Dtype hstart = round(rois_data[2]);
    Dtype wend = round(rois_data[3]);
    Dtype hend = round(rois_data[4]);

    // Check RoI
    if (wstart >= 0 && hstart >= 0 && wstart < wend && hstart < hend &&
        wend < width_im && hend < height_im) {
    } else {
      top_data[rois_index * num_class + cls_id] = kMIN_SCORE;
      // 这里面是for循环，用return会中断后续的循环
      // return;
      continue;
    }

    // caculate the inner and outer RoI coordinate
    Dtype width_roi = wend - wstart;
    Dtype height_roi = hend - hstart;
    Dtype context_scale = 1.8;
    // Dtype context_scale = sqrtf(2.0);
    Dtype width_roi_inner = width_roi / context_scale;
    Dtype height_roi_inner = height_roi / context_scale;
    Dtype width_roi_outer = width_roi * context_scale;
    Dtype height_roi_outer = height_roi * context_scale;
    Dtype wcenter = (wend + wstart) / 2.0;
    Dtype hcenter = (hend + hstart) / 2.0;

    Dtype wstart_inner = wcenter - width_roi_inner / 2.0;
    Dtype hstart_inner = hcenter - height_roi_inner / 2.0;
    Dtype wend_inner = wcenter + width_roi_inner / 2.0;
    Dtype hend_inner = hcenter + height_roi_inner / 2.0;

    Dtype wstart_outer = max(wcenter - width_roi_outer / 2.0, 0.0);
    Dtype hstart_outer = max(hcenter - height_roi_outer / 2.0, 0.0);
    Dtype wend_outer = min(wcenter + width_roi_outer / 2.0, width_im - 1.0);
    Dtype hend_outer = min(hcenter + height_roi_outer / 2.0, height_im - 1.0);

    // assign the coordinate to pixel
    wstart = round(wstart);
    hstart = round(hstart);
    wend = round(wend);
    hend = round(hend);
    wstart_inner = round(wstart_inner);
    hstart_inner = round(hstart_inner);
    wend_inner = round(wend_inner);
    hend_inner = round(hend_inner);
    wstart_outer = round(wstart_outer);
    hstart_outer = round(hstart_outer);
    wend_outer = round(wend_outer);
    hend_outer = round(hend_outer);

    width_roi = wend - wstart + 1;
    height_roi = hend - hstart + 1;
    width_roi_inner = wend_inner - wstart_inner + 1;
    height_roi_inner = hend_inner - hstart_inner + 1;
    width_roi_outer = wend_outer - wstart_outer + 1;
    height_roi_outer = hend_outer - hstart_outer + 1;

    // CPG sum of RoI
    Dtype sum_roi = 0.0;
    // CPG sum of inner RoI
    Dtype sum_inner = 0.0;
    // CPG sum of outer RoI
    Dtype sum_outer = 0.0;
    for (int c = 0; c < channels; ++c) {
      for (int h = hstart_outer; h <= hend_outer; ++h) {
        for (int w = wstart_outer; w <= wend_outer; ++w) {
          int data_index = (c * height_im + h) * width_im + w;
          Dtype g = opg_data[data_index];
          if (g < threshold) {
            continue;
          }

          if (h >= hstart && h <= hend && w >= wstart && w <= wend) {
            /*sum_roi += g;*/
            sum_roi++;
          }

          if (h >= hstart_inner && h <= hend_inner && w >= wstart_inner &&
              w <= wend_inner) {
            /*sum_inner += g;*/
            sum_inner++;
          }

          /*sum_outer += g;*/
          sum_outer++;
        }
      }
    }

    Dtype area_roi = height_roi * width_roi;
    Dtype area_inner = height_roi_inner * width_roi_inner;
    Dtype area_outer = height_roi_outer * width_roi_outer;

    Dtype area_frame = max(area_roi - area_inner, Dtype(1));
    Dtype area_context = max(area_outer - area_roi, Dtype(1));

    //-----------------------------------------------------------------------
    // current best
    Dtype score = (sum_roi - sum_inner) / sqrt(area_frame) -
                  (sum_outer - sum_roi) / sqrt(area_context);

    // bad at test debug
    // Dtype score = (sum_roi - sum_inner) - (sum_outer - sum_roi);

    // (msra 0223):
    // Dtype score = ((sum_roi - 2.0 * (sum_outer - sum_roi)) *
    //(2.0 * (sum_roi - sum_inner) - sum_inner)) /
    // area_roi;
    // if ((sum_roi - 2.0 * (sum_outer - sum_roi)) < 0 &&
    //(2.0 * (sum_roi - sum_inner) - sum_inner) < 0) {
    // score = -1.0 * score;
    //}

    // (msra 0101): bad
    // Dtype score = sqrt((sum_roi - sum_inner) / area_frame) -
    //               sqrt((sum_outer - sum_roi) / area_context);

    // (msra 12.30): very bad
    // Dtype score =
    //    (sum_roi - sum_inner) / area_frame - (sum_outer - sum_roi) /
    // area_context;

    // (msra 12.29): bad
    // Dtype score = ((sum_roi - sum_inner) - (sum_outer - sum_roi)) /
    // area_frame;

    // (msra 0105): bad than (msra 12.29)
    // Dtype score = ((sum_roi - sum_inner) - (sum_outer - sum_roi)) /
    // sqrt(area_frame);

    //-----------------------------------------------------------------------

    // if (sum_roi < min_mass) score = kMIN_SCORE;

    top_data[rois_index * num_class + cls_id] = score;
  }
}

template <typename Dtype>
__global__ void SumBBoxes(const int num_roi, const Dtype *opg_data,
                          const int num, const int channels,
                          const int height_im, const int width_im,
                          const Dtype *rois_data, const int num_class,
                          const int cls_id, const Dtype threshold,
                          const Dtype min_density, const Dtype min_mass,
                          Dtype *const top_data) {
  CUDA_KERNEL_LOOP(index, num_roi) {
    int rois_index = index;

    rois_data += 5 * rois_index;
    int wstart = max(static_cast<int>(round(rois_data[1])), 0);
    int hstart = max(static_cast<int>(round(rois_data[2])), 0);
    int wend = min(static_cast<int>(round(rois_data[3])), width_im);
    int hend = min(static_cast<int>(round(rois_data[4])), height_im);

    Dtype height_roi = hend - hstart + 1;
    Dtype width_roi = wend - wstart + 1;
    Dtype context_scale = static_cast<Dtype>(1.8);
    Dtype height_roi_inner = static_cast<Dtype>(height_roi) / context_scale;
    Dtype width_roi_inner = static_cast<Dtype>(width_roi) / context_scale;
    Dtype height_roi_outer = static_cast<Dtype>(height_roi) * context_scale;
    Dtype width_roi_outer = static_cast<Dtype>(width_roi) * context_scale;
    Dtype hcenter = static_cast<Dtype>(hend + hstart) / 2;
    Dtype wcenter = static_cast<Dtype>(wend + wstart) / 2;

    int wstart_inner =
        max(static_cast<int>(floor(wcenter - width_roi_inner / 2)), 0);
    int hstart_inner =
        max(static_cast<int>(floor(hcenter - height_roi_inner / 2)), 0);
    int wend_inner = min(
        static_cast<int>(ceil(wcenter + width_roi_inner / 2)) + 1, width_im);
    int hend_inner = min(
        static_cast<int>(ceil(hcenter + height_roi_inner / 2)) + 1, height_im);

    int wstart_outer =
        max(static_cast<int>(floor(wcenter - width_roi_outer / 2)), 0);
    int hstart_outer =
        max(static_cast<int>(floor(hcenter - height_roi_outer / 2)), 0);
    int wend_outer = min(
        static_cast<int>(ceil(wcenter + width_roi_outer / 2)) + 1, width_im);
    int hend_outer = min(
        static_cast<int>(ceil(hcenter + height_roi_outer / 2)) + 1, height_im);

    // CPG sum of RoI
    Dtype sum_roi = 0;
    // CPG sum of inner RoI
    Dtype sum_inner = 0;
    // CPG sum of outer RoI
    Dtype sum_outer = 0;
    for (int c = 0; c < channels; ++c) {
      for (int h = hstart_outer; h < hend_outer; ++h) {
        for (int w = wstart_outer; w < wend_outer; ++w) {
          int data_index = (c * height_im + h) * width_im + w;
          Dtype g = opg_data[data_index];
          if (g < threshold) {
            continue;
          }

          if (h > hstart && h < hend && w > wstart && w < wend) {
            /*sum_roi += g;*/
            sum_roi++;
          }

          if (h > hstart_inner && h < hend_inner && w > wstart_inner &&
              w < wend_inner) {
            /*sum_inner += g;*/
            sum_inner++;
          }

          /*sum_outer += g;*/
          sum_outer++;
        }
      }
    }

    Dtype area_roi = height_roi * width_roi;
    Dtype area_inner = height_roi_inner * width_roi_inner;
    Dtype area_outer = height_roi_outer * width_roi_outer;

    Dtype area_frame = max(area_roi - area_inner, Dtype(1));
    Dtype area_context = max(area_outer - area_roi, Dtype(1));

    // current best
    Dtype score = (sum_roi - sum_inner) / sqrt(area_frame) -
                  (sum_outer - sum_roi) / sqrt(area_context);

    // Dtype score = (sum_roi - (sum_outer - sum_roi)) / sqrt(area_roi);
    // Dtype score = (sum_roi - (sum_outer - sum_roi)) / (area_roi);

    // (msra 0101): bad
    // Dtype score = sqrt((sum_roi - sum_inner) / area_frame) -
    //               sqrt((sum_outer - sum_roi) / area_context);

    // (msra 12.30): very bad
    // Dtype score =
    //    (sum_roi - sum_inner) / area_frame - (sum_outer - sum_roi) /
    // area_context;

    // (msra 12.29): bad
    // Dtype score = ((sum_roi - sum_inner) - (sum_outer - sum_roi)) /
    // area_frame;

    // (msra 0105): bad than (msra 12.29)
    // Dtype score = ((sum_roi - sum_inner) - (sum_outer - sum_roi)) /
    // sqrt(area_frame);

    // if (sum_roi < min_mass) score = -1.0;

    top_data[rois_index * num_class + cls_id] = score;
  }
}

template <typename Dtype>
__global__ void IoUBBoxes(const int num_roi, const Dtype *opg_data,
                          const int num, const int channels,
                          const int height_im, const int width_im,
                          const Dtype *rois_data, const int num_class,
                          const int cls_id, const Dtype threshold,
                          const Dtype min_density, const Dtype min_mass,
                          Dtype *const top_data) {
  CUDA_KERNEL_LOOP(index, num_roi) {
    int rois_index = index;

    rois_data += 5 * rois_index;
    int wstart = max(static_cast<int>(round(rois_data[1])), 0);
    int hstart = max(static_cast<int>(round(rois_data[2])), 0);
    int wend = min(static_cast<int>(round(rois_data[3])), width_im);
    int hend = min(static_cast<int>(round(rois_data[4])), height_im);

    Dtype intersections = 0;
    Dtype unions = 0;
    for (int c = 0; c < channels; ++c) {
      for (int h = 0; h < height_im; ++h) {
        bool h_in = (h > hstart && h < hend) ? true : false;
        for (int w = 0; w < width_im; ++w) {
          bool w_in = (w > wstart && w < wend) ? true : false;
          int data_index = (c * height_im + h) * width_im + w;
          Dtype g = opg_data[data_index];

          if (g > threshold || (h_in && w_in)) {
            intersections++;
          }

          if (g > threshold && (h_in && w_in)) {
            unions++;
          }
        }
      }
    }

    top_data[rois_index * num_class + cls_id] = unions / intersections;
  }
}

template <typename Dtype>
bool RepartitionLayer<Dtype>::Need_Order(const int cls_id, const Dtype label,
                                         const Dtype predict) {
  if (cls_id == ignore_label_) return false;
  // assum score is betwween 0 ~ 1
  if (this->phase_ == TRAIN) {
    if (label < 0.5) return false;
    if (is_order_ && predict > predict_order_) {
      return true;
    } else {
      return false;
    }
  } else if (this->phase_ == TEST) {
    return false;
  } else {
    LOG(FATAL) << "unkown phase: " << this->phase_;
  }
  LOG(FATAL) << "We should not arrive here!";
  return false;
}

template <typename Dtype>
void RepartitionLayer<Dtype>::After() {
  // this should not in the Reshape function
  // as Reshape function will be call before start
  pass_im_ += num_im_;

  if (pass_im_ % display_ == 0 && this->phase_ != TEST) {
    if (is_order_) {
      order_threshold_ =
          1.0 - 1.0 * (int(1.0 * pass_im_ / order_step_) + 1) / order_K_;
      if (order_threshold_ < 0) order_threshold_ = 0;
      LOG(INFO) << "#im:" << pass_im_
                << " order_threshold_: " << order_threshold_;
    }
  }
}

template <typename Dtype>
void RepartitionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
                                          const vector<Blob<Dtype> *> &top) {
  if (!is_opg_) {
    caffe_gpu_set(top[0]->count(), Dtype(1), top[0]->mutable_gpu_data());
    return;
  }

  InitFilter(bottom[bottom_index_["label"]]->gpu_data(),
             filter_.mutable_gpu_data());

  LOG_IF(INFO, debug_info_) << "------------------start-----------------------";
  LOG_IF(INFO, debug_info_) << "pass_im_: " << pass_im_;
  //-----------------------------------------------------------------------
  //-----------------------------------------------------------------------
  const Dtype *bottom_label = bottom[bottom_index_["label"]]->cpu_data();
  const Dtype *bottom_predict = bottom[bottom_index_["predict"]]->cpu_data();
  const Dtype *rois_score = bottom[bottom_index_["rois_score"]]->cpu_data();

  Dtype *pos_label_data;
  Dtype *neg_label_data;
  if (is_order_ && top.size() == 3) {
    pos_label_data = top[1]->mutable_cpu_data();
    neg_label_data = top[2]->mutable_cpu_data();
  }

  int re_num = 0;
  for (int cls_id = 0; cls_id < num_class_; ++cls_id) {
    int index = cls_id;
    LOG_IF(INFO, debug_info_) << "class: " << voc_label_[cls_id]
                              << "\t\tlabel: " << bottom_label[index]
                              << " score: " << bottom_predict[index];

    // whether need repartition
    // TODO(YH): we need find more efficient way to do this
    Dtype asum;
    caffe_gpu_asum(opg_size_,
                   bottom[bottom_index_["opg"]]->gpu_data() +
                       bottom[bottom_index_["opg"]]->offset(0, cls_id, 0, 0),
                   &asum);
    if (asum == 0) continue;
    //-----------------------------------------------------------------------
    // propocess data
    switch (this->layer_param_.cpg_param().mode()) {
      case CPGParameter_Mode_DEFAULT:
      case CPGParameter_Mode_PRED:
      case CPGParameter_Mode_CPG_POOLING: {
        caffe_gpu_set(raw_data_->count(), Dtype(0),
                      raw_data_->mutable_gpu_data());
        caffe_gpu_set(raw_data_->count(), Dtype(0),
                      raw_data_->mutable_gpu_diff());
        caffe_gpu_abs(opg_size_,
                      bottom[bottom_index_["opg"]]->gpu_data() +
                          bottom[bottom_index_["opg"]]->offset(0, cls_id, 0, 0),
                      raw_data_->mutable_gpu_data());

        // TODO(YH): order_threshold_
        if (order_threshold_ > 0 &&
            Need_Order(cls_id, bottom_label[index], bottom_predict[index])) {
          caffe_cpu_threshold_bbox(raw_data_, bboxes_, fg_threshold_, cls_id);
          Dtype max_size = 0;
          const Dtype *bbox = bboxes_->cpu_data();
          for (int box_id = 0; box_id < max_bb_per_cls_; ++box_id) {
            if (bbox[0] == -1) break;
            Dtype size = (bbox[2] - bbox[0]) * (bbox[3] - bbox[1]);
            if (max_size < size) max_size = size;
            bbox += bboxes_->offset(1);
          }
          Dtype max_scale = max_size / (height_im_ * width_im_);
          if (max_scale < order_threshold_) {
            pos_label_data[cls_id] = -1;
            neg_label_data[cls_id] = -1;
            continue;
          }
        }
      } break;
      case CPGParameter_Mode_CRF:
        break;
      default:
        LOG(FATAL) << "Unknown mode.";
    }

    ++re_num;

    switch (this->layer_param_.cpg_param().mode()) {
      case CPGParameter_Mode_DEFAULT: {
        const Dtype maxval =
            caffe_cpu_max_element(opg_size_, raw_data_->cpu_data());
        const Dtype threshold = maxval * fg_threshold_;

        // NOLINT_NEXT_LINE(whitespace/operators)
        get_above_mask<Dtype> << <CAFFE_GET_BLOCKS(opg_size_),
                                  CAFFE_CUDA_NUM_THREADS>>>
            (opg_size_, raw_data_->gpu_data(), raw_data_->mutable_gpu_diff(),
             threshold);
        Dtype im_mass;
        caffe_gpu_asum(opg_size_, raw_data_->gpu_diff(), &im_mass);
        const Dtype im_density = 1.0 * im_mass / height_im_ / width_im_;

        LOG_IF(INFO, debug_info_) << "maxval: " << maxval
                                  << " threshold: " << threshold
                                  << " im_mass: " << im_mass
                                  << " im_density: " << im_density;
        LOG_IF(INFO, debug_info_) << "WeightBBoxes:";

        // NOLINT_NEXT_LINE(whitespace/operators)
        WeightBBoxes<Dtype> << <CAFFE_GET_BLOCKS(num_roi_),
                                CAFFE_CUDA_NUM_THREADS>>>
            (num_roi_, raw_data_->gpu_data(), 1, 1, height_im_, width_im_,
             bottom[bottom_index_["rois"]]->gpu_data(), num_class_, cls_id,
             threshold, im_density * density_threshold_,
             im_mass * mass_threshold_, filter_.mutable_gpu_data());
      } break;
      case CPGParameter_Mode_PRED: {
        const Dtype maxval =
            caffe_cpu_max_element(opg_size_, raw_data_->cpu_data());
        const Dtype threshold = maxval * fg_threshold_;

        // NOLINT_NEXT_LINE(whitespace/operators)
        get_above_mask<Dtype> << <CAFFE_GET_BLOCKS(opg_size_),
                                  CAFFE_CUDA_NUM_THREADS>>>
            (opg_size_, raw_data_->gpu_data(), raw_data_->mutable_gpu_diff(),
             threshold);
        Dtype im_mass;
        caffe_gpu_asum(opg_size_, raw_data_->gpu_diff(), &im_mass);
        const Dtype im_density = 1.0 * im_mass / height_im_ / width_im_;

        LOG_IF(INFO, debug_info_) << "maxval: " << maxval
                                  << " threshold: " << threshold
                                  << " im_mass: " << im_mass
                                  << " im_density: " << im_density;
        LOG_IF(INFO, debug_info_) << "ScoreBBoxes:";
        const Dtype min_density = im_density * density_threshold_;
        // NOLINT_NEXT_LINE(whitespace/operators)
        ScoreBBoxes<Dtype> << <CAFFE_GET_BLOCKS(num_roi_),
                               CAFFE_CUDA_NUM_THREADS>>>
            (num_roi_, raw_data_->gpu_data(), 1, 1, height_im_, width_im_,
             bottom[bottom_index_["rois"]]->gpu_data(), num_class_, cls_id,
             threshold, min_density, im_mass, filter_.mutable_gpu_data());
      } break;
      case CPGParameter_Mode_CPG_POOLING: {
        int max_value_index;
        caffe_gpu_amax(opg_size_, raw_data_->gpu_data(), &max_value_index);
        max_value_index--;
        const Dtype maxval = raw_data_->cpu_data()[max_value_index];
        const Dtype threshold = maxval * fg_threshold_;

        // NOLINT_NEXT_LINE(whitespace/operators)
        get_above_mask<Dtype> << <CAFFE_GET_BLOCKS(opg_size_),
                                  CAFFE_CUDA_NUM_THREADS>>>
            (opg_size_, raw_data_->gpu_data(), raw_data_->mutable_gpu_diff(),
             threshold);
        Dtype im_mass;
        caffe_gpu_asum(opg_size_, raw_data_->gpu_diff(), &im_mass);
        const Dtype im_density = 1.0 * im_mass / height_im_ / width_im_;

        // CHECK_GE(maxval, 0) << "maxval should be greater than 0.";
        LOG_IF(INFO, debug_info_) << "maxval: " << maxval
                                  << " threshold: " << threshold
                                  << " im_mass: " << im_mass
                                  << " im_density: " << im_density;
        LOG_IF(INFO, debug_info_) << "SumBBoxes:";

        // NOLINT_NEXT_LINE(whitespace/operators)
        CPGPooling<Dtype> << <CAFFE_GET_BLOCKS(num_roi_),
                              CAFFE_CUDA_NUM_THREADS>>>
            (num_roi_, raw_data_->gpu_data(), 1, 1, height_im_, width_im_,
             bottom[bottom_index_["rois"]]->gpu_data(), num_class_, cls_id,
             threshold, im_density * density_threshold_,
             im_mass * mass_threshold_, filter_.mutable_gpu_data());

        Dtype re_predict = 0;

        // normalization max value to |1|
        if (true) {
          Dtype *filter_data = filter_.mutable_cpu_data();
          Dtype max_value = 0;
          Dtype min_value = 0;
          for (int roi_id = 0; roi_id < num_roi_; roi_id++) {
            Dtype value = filter_data[roi_id * num_class_ + cls_id];
            if (value > max_value) {
              max_value = value;
            }
            if (value < min_value && value != kMIN_SCORE) {
              min_value = value;
            }
          }
          // CHECK_GE(max_value, 0) << "max_value should be greater than 0.";
          // CHECK_GE(min_value, -1) << "min_value should be -1.";
          if (max_value > 0 && min_value < 0) {
            for (int roi_id = 0; roi_id < num_roi_; roi_id++) {
              Dtype value = filter_data[roi_id * num_class_ + cls_id];
              if (value == kMIN_SCORE) {
                value = -1;
              } else {
                value = value > 0 ? value / max_value : value / (-min_value);
              }
              // value = value > 0 ? value / max_value : -1;
              filter_data[roi_id * num_class_ + cls_id] = value;

              re_predict +=
                  value > 0 ? value * rois_score[roi_id * num_class_ + cls_id]
                            : 0;
            }
          } else if (max_value > 0 && min_value == 0) {
            for (int roi_id = 0; roi_id < num_roi_; roi_id++) {
              Dtype value = filter_data[roi_id * num_class_ + cls_id];
              if (value == kMIN_SCORE) {
                value = -1;
              } else {
                value = value / max_value;
              }
              filter_data[roi_id * num_class_ + cls_id] = value;

              re_predict += value * rois_score[roi_id * num_class_ + cls_id];
            }
          } else {
            for (int roi_id = 0; roi_id < num_roi_; roi_id++) {
              filter_data[roi_id * num_class_ + cls_id] = 1.0;
              re_predict += 1 * rois_score[roi_id * num_class_ + cls_id];
            }
          }
        }

        // drop
        if (false) {
          if (this->phase_ == TRAIN) {
            double secret;
            caffe_rng_uniform(1, 0.0, 1.0, &secret);

            if (secret < min(max(bottom_predict[index] - re_predict, 0.0) +
                                 1.0 * pass_im_ / (5011 * 2 * 20),
                             1.0)) {
              Dtype *filter_data = filter_.mutable_cpu_data();
              for (int roi_id = 0; roi_id < num_roi_; roi_id++) {
                filter_data[roi_id * num_class_ + cls_id] = 1;
              }
            }
          }
        }

        if (debug_info_) {
          const Dtype *filter_data = filter_.cpu_data();
          const Dtype *rois_data = bottom[bottom_index_["rois"]]->cpu_data();
          int a0 = 0;
          int b0 = 0;
          int e0 = 0;
          for (int roi_id = 0; roi_id < num_roi_; roi_id++) {
            Dtype value = filter_data[roi_id * num_class_ + cls_id];
            std::cout << value << "(" << rois_data[roi_id * 5 + 1] << " "
                      << rois_data[roi_id * 5 + 2] << " "
                      << rois_data[roi_id * 5 + 3] << " "
                      << rois_data[roi_id * 5 + 4] << ") ";
            if (value > 0)
              a0++;
            else if (value < 0)
              b0++;
            else
              e0++;
          }
          std::cout << std::endl;
          std::cout << "a0: " << a0 << " b0: " << b0 << " e0: " << e0
                    << std::endl;
          std::cout << "re_predict: " << re_predict << std::endl;
        }
      } break;
      case CPGParameter_Mode_CRF:
        break;
      default:
        LOG(FATAL) << "Unknown mode.";
    }
  }

  //----------------------------------------------------------------------
  // Show patch
  //----------------------------------------------------------------------
  if (debug_info_) {
    Show_rois(bottom[bottom_index_["rois"]], &filter_,
              bottom[bottom_index_["label"]], pass_im_, num_im_, voc_label_,
              ignore_label_, predict_threshold_);
  }

  // get the final output from filter
  switch (this->layer_param_.cpg_param().mode()) {
    case CPGParameter_Mode_DEFAULT:
    case CPGParameter_Mode_PRED:
      top[0]->CopyFrom(filter_, false, false);
      if (bottom.size() == bottom_index_["io"] + 1) {
        if (re_num > 0) {
          int save_id = int(bottom[bottom_index_["io"]]->cpu_data()[0]);
          LOG_IF(INFO, debug_info_) << "save_id: " << save_id;

          BlobProto save_blob;
          top[0]->ToProto(&save_blob, false);
          stringstream save_path;
          save_path << "data/opg_cache/" << save_id;
          WriteProtoToBinaryFile(save_blob, save_path.str());
        }

        caffe_gpu_or(num_roi_ * num_class_,
                     bottom[bottom_index_["filter"]]->gpu_data(),
                     top[0]->gpu_data(), top[0]->mutable_gpu_data());
      }
      break;
    case CPGParameter_Mode_CPG_POOLING:
      top[0]->CopyFrom(filter_, false, false);
      if (bottom.size() == bottom_index_["io"] + 1) {
        if (re_num > 0) {
          int save_id = int(bottom[bottom_index_["io"]]->cpu_data()[0]);
          LOG_IF(INFO, debug_info_) << "save_id: " << save_id;

          BlobProto save_blob;
          top[0]->ToProto(&save_blob, false);
          stringstream save_path;
          save_path << "data/opg_cache/" << save_id;
          WriteProtoToBinaryFile(save_blob, save_path.str());
        }
        caffe_gpu_maximum(num_roi_ * num_class_,
                          bottom[bottom_index_["filter"]]->gpu_data(),
                          top[0]->gpu_data(), top[0]->mutable_gpu_data());

        // caffe_gpu_add(num_roi_ * num_class_,
        // bottom[bottom_index_["filter"]]->gpu_data(),
        // top[0]->gpu_data(), top[0]->mutable_gpu_data());
        // caffe_gpu_threshold(num_roi_ * num_class_, top[0]->gpu_data(),
        // top[0]->mutable_gpu_data(), Dtype(1), true);
      }
      break;
    case CPGParameter_Mode_CRF:
      break;
    default:
      LOG(FATAL) << "Unknown mode.";
  }

  //-----------------------------------------------------------------------
  LOG_IF(INFO, debug_info_) << " top: " << top[0]->asum_data();

  After();
}

template <typename Dtype>
void RepartitionLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype> *> &top, const vector<bool> &propagate_down,
    const vector<Blob<Dtype> *> &bottom) {
  for (size_t i = 0; i < bottom.size(); i++) {
    if (propagate_down[i]) {
      caffe_gpu_set(bottom[i]->count(), static_cast<Dtype>(0),
                    bottom[i]->mutable_gpu_diff());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(RepartitionLayer);

}  // namespace caffe

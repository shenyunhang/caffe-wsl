#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/pooling_sigmoid_layer.hpp"

namespace caffe {

	template <typename Dtype>
		__global__ void PoolingSigmoidForward(const int n, const Dtype* in, Dtype* out, const Dtype* fliter) {
			CUDA_KERNEL_LOOP(index, n) {
				out[index] = 1. / (1. + exp(-in[index])) * fliter[index];
			}
		}

	template <typename Dtype>
		void PoolingSigmoidLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
				const vector<Blob<Dtype>*>& top) {
			const Dtype* bottom_data = bottom[0]->gpu_data();
			Dtype* top_data = top[0]->mutable_gpu_data();
			const int count = bottom[0]->count();
			const Dtype* fliter_data=bottom[1]->gpu_data();
			// NOLINT_NEXT_LINE(whitespace/operators)
			PoolingSigmoidForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
					count, bottom_data, top_data, fliter_data);
			CUDA_POST_KERNEL_CHECK;
			// << " count: " << count << " bottom_data: "
			//     << (unsigned long)bottom_data
			//     << " top_data: " << (unsigned long)top_data
			//     << " blocks: " << CAFFE_GET_BLOCKS(count)
			//     << " threads: " << CAFFE_CUDA_NUM_THREADS;
		}

	template <typename Dtype>
		__global__ void PoolingSigmoidBackward(const int n, const Dtype* in_diff,
				const Dtype* out_data, Dtype* out_diff, const Dtype* fliter) {
			CUDA_KERNEL_LOOP(index, n) {
				const Dtype sigmoid_x = out_data[index];
				out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x) * fliter[index];
			}
		}

	template <typename Dtype>
		void PoolingSigmoidLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
				const vector<bool>& propagate_down,
				const vector<Blob<Dtype>*>& bottom) {
			if (propagate_down[0]) {
				const Dtype* top_data = top[0]->gpu_data();
				const Dtype* top_diff = top[0]->gpu_diff();
				Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
				const int count = bottom[0]->count();
				const Dtype* fliter_data=bottom[1]->gpu_data();
				// NOLINT_NEXT_LINE(whitespace/operators)
				PoolingSigmoidBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
						count, top_diff, top_data, bottom_diff, fliter_data);
				CUDA_POST_KERNEL_CHECK;
			}
		}

	INSTANTIATE_LAYER_GPU_FUNCS(PoolingSigmoidLayer);


}  // namespace caffe

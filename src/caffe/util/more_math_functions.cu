#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/more_math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void maximum_kernel(const int N, const Dtype* const X,
                               Dtype* const Y, const int s, const int e) {
  CUDA_KERNEL_LOOP(index, N) {
    Y[index] = X[index];
    for (int i = index + s; i < e; i += s) {
      if (Y[index] < X[i]) {
        Y[i] = X[i];
      }
    }
  }
}

template <typename Dtype>
void caffe_gpu_maximum(const int N, const Dtype* const X, Dtype* const Y,
                       const int s, const int e) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  maximum_kernel<Dtype> << <CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>
      (N, X, Y, s, e);
}

template void caffe_gpu_maximum<int>(const int N, const int* const X,
                                     int* const Y, const int s, const int e);
template void caffe_gpu_maximum<float>(const int N, const float* const X,
                                       float* const Y, const int s,
                                       const int e);
template void caffe_gpu_maximum<double>(const int N, const double* const X,
                                        double* const Y, const int s,
                                        const int e);

template <typename Dtype>
__global__ void or_kernel(const int N, const Dtype* const x,
                          const Dtype* const y, Dtype* const z) {
  CUDA_KERNEL_LOOP(index, N) {
    if (x[index] == 1 || y[index] == 1)
      z[index] = 1;
    else
      z[index] = 0;
  }
}

template <typename Dtype>
void caffe_gpu_or(const int N, const Dtype* const x, const Dtype* const y,
                  Dtype* const z) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  or_kernel<Dtype> << <CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>
      (N, x, y, z);
}

template void caffe_gpu_or<int>(const int N, const int* const x,
                                const int* const y, int* z);
template void caffe_gpu_or<float>(const int N, const float* const x,
                                  const float* const y, float* z);
template void caffe_gpu_or<double>(const int N, const double* const x,
                                   const double* const y, double* z);

}  // namespace caffe

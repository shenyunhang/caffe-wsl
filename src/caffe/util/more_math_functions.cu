#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/more_math_functions.hpp"

namespace caffe {

template <typename Dtype>
void caffe_gpu_or(const int N, const Dtype* x, const Dtype* y, Dtype* z) {}

template void caffe_gpu_or<int>(const int N, const int* x, const int* y,
                                int* z);
template void caffe_gpu_or<float>(const int N, const float* x, const float* y,
                                  float* z);
template void caffe_gpu_or<double>(const int N, const double* x,
                                   const double* y, double* z);

template <typename Dtype>
__global__ void maximum_kernel(const int N, const Dtype* const X,
                               Dtype* const Y, const int s, const int e) {
  CUDA_KERNEL_LOOP(index, N) {
    Y[index] = X[index];
    for (int i = index + s; i < e; i += s) {
      if (Y[index] < X[i]) {
        Y[i] = X[i];
      }
    }
  }
}

template <typename Dtype>
void caffe_gpu_maximum(const int N, const Dtype* const X, Dtype* const Y,
                       const int s, const int e) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  maximum_kernel<Dtype> << <CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>
      (N, X, Y, s, e);
}

template void caffe_gpu_maximum(const int N, const int* const X,
                                int* const Y, const int s, const int e);
template void caffe_gpu_maximum(const int N, const float* const X,
                                float* const Y, const int s, const int e);
template void caffe_gpu_maximum(const int N, const double* const X,
                                double* const Y, const int s, const int e);

}  // namespace caffe
